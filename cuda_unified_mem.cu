#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h>
#include "lodepng.h"
#include <vector>
#include <iostream>
#include <chrono>
using namespace std;
using namespace chrono;
typedef unsigned char uchar;

__global__ void make_gray(const uchar *r, const uchar *g, const uchar *b, size_t num_pixels, uchar *gray) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i > num_pixels) {
		return;
	}

	double red_val = 0.2125 * r[i];
	double green_val = 0.7154 * g[i];
	double blue_val = 0.0721 * b[i];

	double gray_val = red_val + green_val + blue_val;

	gray[i] = gray_val;
}

int main()
{
	time_point<steady_clock> time_a = high_resolution_clock::now();
	vector<uchar> image;
	unsigned width, height;
	string image_filename = "test_image.png";
	unsigned exit_code = lodepng::decode(image, width, height, image_filename);

	if (exit_code != 0) {
		cout << "Error opening image " << image_filename << " with the code " << exit_code << ": " << lodepng_error_text(exit_code) << endl;
		exit(exit_code);
	}
	else {
		cout << "Image loaded, size: " << image.size() << " bytes" << endl;
	}
	// Pixels are in 1-D vector, 4 bytes, 4 channels, RGBARGBARGBA

	vector<uchar> red;
	vector<uchar> green;
	vector<uchar> blue;

	for (size_t i = 0; i < image.size(); i += 4) {
		red.push_back(image[i]);
		green.push_back(image[i + 1]);
		blue.push_back(image[i + 2]);
	}

	cout << "Finished preparing channels" << endl;

	size_t num_pixels = red.size();
	int threads_per_block = 512;
	int total_blocks = ((num_pixels + threads_per_block - 1) / threads_per_block);

	uchar *d_red, *d_green, *d_blue, *d_gray;

	hipMallocManaged(&d_red, num_pixels);
	hipMallocManaged(&d_green, num_pixels);
	hipMallocManaged(&d_blue, num_pixels);
	hipMallocManaged(&d_gray, num_pixels);

	hipMemcpy(d_red, red.data(), num_pixels, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(d_green, green.data(), num_pixels, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(d_blue, blue.data(), num_pixels, hipMemcpyKind::hipMemcpyHostToDevice);

	time_point<steady_clock> time_b = high_resolution_clock::now();

	make_gray<<<total_blocks, threads_per_block>>>(d_red, d_green, d_blue, num_pixels, d_gray);

	time_point<steady_clock> time_c = high_resolution_clock::now();

	hipDeviceSynchronize();

	hipFree(d_red);
	hipFree(d_green);
	hipFree(d_blue);

	cout << "Finished creating image" << endl;

	string out_filename = "result_image_cuda_um.png";
	exit_code = lodepng::encode(out_filename, d_gray, width, height, LCT_GREY);
	if (exit_code != 0) {
		cout << "Error saving file " << out_filename << " with the code " << exit_code << ": " << lodepng_error_text(exit_code) << endl;
		exit(exit_code);
	}
	else {
		cout << "Successfully saved file " << out_filename << endl;
	}

	hipFree(d_gray);

	time_point<steady_clock> time_d = high_resolution_clock::now();

	cout << "TOTAL TIME: " << duration_cast<microseconds>(time_d - time_a).count() << " mcs." << endl;
	cout << "CALCULATION TIME: " << duration_cast<microseconds>(time_c - time_b).count() << " mcs." << endl;
}
